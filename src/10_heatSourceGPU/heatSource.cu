#include "hip/hip_runtime.h"
/*
 * @Author: Bingyang Jin
 * @Date: 2022-10-26 20:35:07
 * @Editor: Bingyang Jin
 * @FilePath: /src/10_heatSourceGPU/heatSource.cu
 * @Description: create the file
 */

#include <stdio.h>
#include <vector>
#include <hip/hip_runtime.h>
#include "cuda-samples-master/Common/hip/hip_runtime_api.h"
#include ""
#include "hipblas.h"
#include "hip/device_functions.h"
#include "../common_book/book.h"

#define REAL double
#define MAX_NUMBER 100000
#define ITER_TIME 1000
#define SPEED 0.33f

std::vector<std::vector<int>> adjoinTriangles; // �����ε��ڽ�������
extern std::vector<REAL> gIntensity[2];
extern int currentPass; // ���������ʾ��ǰ gIntensity �ǵڼ������� 0 �� 1 ֮�佻��
extern std::vector<int> gSources; // ��Դ��

struct TriangleAdjoin {
	bool isConstant = false;
	REAL intensity = 0;
	int adj[3];
};

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__global__ void heatCalculate(TriangleAdjoin* d, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		for (int m = 0; m < ITER_TIME; m++) {
			REAL out = d[i].intensity;
			for (int j = 0; j < 3; j++) {
				int index = d[i].adj[j];
				out += SPEED * (d[index].intensity - d[i].intensity);
			}

			__syncthreads();

			d[i].intensity = out;
			if (d[i].isConstant) {
				d[i].intensity = 1;
			}

			__syncthreads();
		}
	}
};

extern "C" int doPropogateGPU()
{
	// ������ʱ��
	hipEvent_t start, stop;
	float elapsedTime;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = gIntensity[currentPass].size();
	int prevPass = currentPass;
	currentPass = 1 - currentPass;

	size_t size = numElements * sizeof(TriangleAdjoin);

	// ׼�� CPU ����
	// Allocate the host input vector h
	TriangleAdjoin* h = (TriangleAdjoin*)malloc(size);

	// Verify that allocations succeeded
	if (h == NULL)
	{
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		//h[i] = gIntensity[prevPass][i];
		h[i].intensity = gIntensity[prevPass][i];
		h[i].isConstant = false;
		for (int j = 0; j < 3; j++) {
			h[i].adj[j] = adjoinTriangles[i][j];
		}
	}
	for (int i = 0; i < gSources.size(); i++) {
		h[gSources[i]].isConstant = true;
	}

	// ��GPU�ϴ����ڴ�
	// Allocate the device input vector d
	TriangleAdjoin* d = NULL;
	err = hipMalloc((void**)&d, size);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device vector d_0 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// �� CPU �ڵ�ֵ������ GPU ��
	// Copy the host input vectors A and B in host memory to the device input vectors in device memory
	err = hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// ���� GPU ����
	// Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 1024;
	int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

	heatCalculate << <blocksPerGrid, threadsPerBlock >> > (d, numElements);

	err = hipGetLastError();

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch heatAdd kernel (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// �� GPU �Ľ�������� CPU ��
	// Copy the device result vector in device memory to the host result vector
	// in host memory.
	free(h);
	h = (TriangleAdjoin*)malloc(size);
	err = hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	for (int i = 0; i < numElements; i++) {
		gIntensity[currentPass][i] = h[i].intensity;
	}
	for (int i = 0; i < gSources.size(); i++) {
		gIntensity[currentPass][gSources[i]] = 1.0;
	}

	//printf("###################\n");

	// �ͷſռ�
	// Free device global memory
	err = hipFree(d);

	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Free host memory
	free(h);

	// ����ʱ��
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("Do %d Times and Time taken: %3.1f ms\n", ITER_TIME, elapsedTime);

	return 0;
}